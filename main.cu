/******************************************************************************

ROLL NUMBER : IEC2019065
NAME        : JAYA MUKESH G

DESCRIPTION : Program to multipl sparse matrix and Dense Vector with and without using threads

SUMMARY     : First the COO file is read and converted into CSR format with matrix_to_csr() function.
              Then matrix-vector multiplication is performed using threads by thread_multiplication() 
              function and result is printed. Then matrix-vector multiplication is performed without 
              using threads by multiplication() function and result is printed.

OUTPUT      :   1) A-vector in CSR and 2d formats.
                2) C-vector and time taken by each method

OS          : Ubuntu
COMPILER    : g++
COMMAND     : g++ -std=c++11 -pthread iec2019065.cpp

*******************************************************************************/


#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
using namespace std;
using namespace std::chrono;

# define NUM_THREADS 4

// thread parameters
int num_of_threads;
int thread_count_track =0;

// Matrix A components for CSR format
vector<double>csr_val, csr_row, csr_col;


//Matrix B and C
vector<double>b_vector;
vector<double>c_vector;
int vector_size;


// function to convert 2d sparse matrix to CSR format
void matrix_to_csr(vector<vector<double>>&matrix)
{
    int temp_count = 0;

    for(int i=0;i<matrix.size();i++)
    {
        for(int j=0;j<matrix[i].size();j++)
        {
            if(matrix[i][j] != 0)
            {
                csr_val.push_back(matrix[i][j]);
                csr_col.push_back(j);
                temp_count++;
            }
        }

        csr_row.push_back(temp_count);
    }
}

// function to perform CSR-vector multiplication without using threads
void multiplication()
{
    for(int i=0;i<vector_size;i++)
    {
        int position = 0;
        if(i) position = csr_row[i-1];
        for(int j=position; j<csr_row[i] ; j++)
        {
            c_vector[i] += csr_val[j] * b_vector[csr_col[j]];
        }
    }
}

// function to perform CSR-vector multiplication by using threads
__global__
void thread_multiplication(double *c_vector_cuda, double *csr_row_cuda, double *csr_col_cuda, double *csr_val_cuda, double *b_vector_cuda, double vector_size_cuda)
{
    // find the batch to be processed by the current thread
    int batch_start = threadIdx.x;
    int stride = blockDim.x;

    for( int i = batch_start;  i<vector_size_cuda; i += stride )
    {
        int position = 0;
        if(i) position = csr_row_cuda[i-1];
        for(int j=position; j<csr_row_cuda[i] ; j++)
        {
            int temp = csr_col_cuda[j];
            c_vector_cuda[i] += csr_val_cuda[j] * (b_vector_cuda[temp]);
        }
    }

    printf("\nCUDA\n");
}

int main()
{
    // read COO file
    ifstream file("inputfile.mtx");
    int M, N, L;
    while (file.peek() == '%') file.ignore(2048, '\n');
    file >> M>> N >> L;

    vector_size = M;

        // 2d matrix declaration
    vector<vector<double>> matrix(M, vector<double>(N, 0.0));
    
        // fill the matrix with non zero values while reading COO file
    for (int l = 0; l < L; l++)
    {
        double data;
        int csr_row, csr_col;
        file >> csr_row >> csr_col >> data;
        matrix[csr_row -1][csr_col -1] = data;
    }
        
    file.close();
    
    // read vector.txt file
    fstream fn;
    string word, filename;
    filename = "vector.txt";
    fn.open(filename.c_str());
    while (fn >> word)
    {
        int x = stoi(word);
        b_vector.push_back(x);
    }
    
    // covert 2d matrix to CSR format
    matrix_to_csr(matrix);
    
    // cout<<"\nCSR Value vector: \n";
    // for(auto i:csr_val) cout<<i<<" "; cout<<"\n";
    // cout<<"\nCSR Column vector: \n";
    // for(auto i:csr_col) cout<<i<<" "; cout<<"\n";
    // cout<<"\nCSR Row vector: \n";
    // for(auto i:csr_row) cout<<i<<" "; cout<<"\n";

    // print matrix A
    // for(int i=0;i<M;i++)
    // {
    //     for(int j=0;j<N;j++)
    //     {
    //         cout<<matrix[i][j]<<" ";
    //     }
    //     cout<<endl;
    // }

    cout<<endl;
    
    
    // multiplication with using threads
    num_of_threads = NUM_THREADS;
    // cout<<"\nMultiplication Using Threads\n";

    
    
    // vector<thread> threads(num_of_threads);
    // c_vector.assign(vector_size, 0);

    //     // START TIMER
    // auto start = high_resolution_clock::now();

    // for(int i=0;i<num_of_threads;i++)
    // threads[i] = thread(thread_multiplication);

    // for(int i=0;i<num_of_threads;i++)
    // threads[i].join();

    //     // STOP TIMER AND PRINT DURATION
    // auto stop = high_resolution_clock::now();
    // auto duration = duration_cast<microseconds>(stop - start);
    // cout<< "\nTime taken : " << duration.count() << " microseconds\n" << endl;


    //CUDA 

    cout<<b_vector.size()<<endl;


    

        //mem allocation
    double *c_vector_cuda_temp, *csr_row_cuda_temp, *csr_col_cuda_temp, *csr_val_cuda_temp, *b_vector_cuda_temp;

   
        //c_vector
    hipMallocManaged(&c_vector_cuda_temp, vector_size*sizeof(double));

    hipMallocManaged(&csr_row_cuda_temp, csr_row.size()*sizeof(double));
    

    for(auto i=0; i<csr_row.size(); i++)
    csr_row_cuda_temp[i] = csr_row[i];


    hipMallocManaged(&csr_col_cuda_temp, csr_col.size()*sizeof(double));
    for(auto i=0; i<csr_col.size(); i++)
    csr_col_cuda_temp[i] = csr_col[i];

    
    

    

    hipMallocManaged(&csr_val_cuda_temp, csr_val.size()*sizeof(double));
    for(auto i=0; i<csr_val.size(); i++)
    csr_val_cuda_temp[i] = csr_val[i];

    

    
    hipMallocManaged(&b_vector_cuda_temp, vector_size*sizeof(double));
    for(auto i=0; i<b_vector.size(); i++)
    b_vector_cuda_temp[i] = b_vector[i];


    

    

    for(auto i:b_vector)
    cout<<i<<" ";
    cout<<endl;

    for(int i=0; i<vector_size; i++)
    printf("%f ",b_vector_cuda_temp[i]);
    printf("\n");

    printf("\nCUDA\n");

    thread_multiplication<<<1,NUM_THREADS>>>(
        c_vector_cuda_temp,
        csr_row_cuda_temp,
        csr_col_cuda_temp,
        csr_val_cuda_temp,
        b_vector_cuda_temp,
        vector_size
    );

    hipDeviceSynchronize();

    cout<<"\n\n\n";

    for(int i=0; i<vector_size; i++)
    cout<<c_vector_cuda_temp[i]<<" ";



    //     // PRINT C_VECTOR
    // for(auto i:c_vector) 
    // cout<<i<<" "; 
    // cout<<"\n";
    
    // // multiplication without using threads
    // cout<<"\nMultiplication Without Using Threads : \n";
    
    
    // c_vector.assign(vector_size, 0);

    //     // START TIMER
    // start = high_resolution_clock::now();
    // multiplication();
    //     // STOP TIMER AND PRINT DURATION
    // stop = high_resolution_clock::now();
    // duration = duration_cast<microseconds>(stop - start);
    // cout<< "\nTime taken : " << duration.count() << " microseconds\n" << endl;


    //     // PRINT C_VECTOR
    // for(auto i:c_vector) 
    // cout<<i<<" "; 
    // cout<<"\n";
    
    return 0;
}
